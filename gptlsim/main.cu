#include <stdio.h>
#include <hip/hip_runtime.h>
#include "./proto.h"

int main ()
{
  Timer *table_cpu;
  size_t nbytes;  // number of bytes to allocate
  int gridsize, blocksize;
  int maxwarps;

  printf ("Enter gridsize\n");
  (void) scanf ("%d", &gridsize);

  printf ("Enter blocksize\n");
  (void) scanf ("%d", &blocksize);

  printf ("Enter maxwarps (gridsize*blocksize=%d)\n", gridsize*blocksize);
  (void) scanf ("%d", &maxwarps);

  nbytes = maxwarps * sizeof (Timer);
  gpuErrchk (hipMalloc (&table_cpu, nbytes));

  init_sim <<<1,1>>> (table_cpu, maxwarps);
  hipDeviceSynchronize ();
  
  run_sim <<<gridsize,blocksize>>> ();
  hipDeviceSynchronize ();
  return 0;
}
